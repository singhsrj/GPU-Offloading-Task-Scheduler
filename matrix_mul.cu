/*nvcc -O2 -lineinfo -arch=sm_86 -o matrix_mul matrix_mul.cu
 ls -l matrix_mul
 ./matrix_mul
nsys profile -o matrix_mul_report ./matrix_mul
nsys export --report cudaapisum --output matrix_mul_summary.csv --force matrix_mul_report.qdrep
nsys-ui
source gpu_env/bin/activate
 */
#include <iostream>
#include <hip/hip_runtime.h>

#define N 512  // Matrix size (N x N)

// CUDA kernel for matrix multiplication
__global__ void matrixMul(const float *A, const float *B, float *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;
    if (row < n && col < n) {
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    size_t bytes = N * N * sizeof(float);

    // Allocate host memory
    float *h_A = new float[N * N];
    float *h_B = new float[N * N];
    float *h_C = new float[N * N];

    // Initialize input matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Copy data to device
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // Configure grid and block dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);

    // Launch kernel
    matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Copy results back
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    std::cout << "Matrix multiplication completed successfully.\n";

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
